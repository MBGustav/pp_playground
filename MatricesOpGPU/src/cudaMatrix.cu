#include "hip/hip_runtime.h"
#include <iostream>
#include "cudaMatrix.cuh"

// template<typename TT>
__host__ void cudaMatrix::allocate(int row,int col)
{
    this->_row = row;
    this->_col = col;
    // to make padding efficient
    this->_ld  = ld_padding(col);

    // TODO: should I implement here te Async Memory, or alloc through the op? No..
    
    MALLOC(host_data, this->MemSize(true));

    hipMalloc(&dev_data, this->MemSize(true));

    check_last_error();
    is_transposed=false;
    Change = Equal;
}

// template<typename TT>
__host__ void cudaMatrix::deallocate()
{
    free(host_data);
    hipFree(dev_data);
    check_last_error ();
}

// template<typename TT>
__host__ cudaMatrix::cudaMatrix() : host_data{NULL}, dev_data{NULL}{}

// template<typename TT>
__host__ cudaMatrix::cudaMatrix(int row, int col) : _row(row), _col(col){allocate(row, col);}

// template<typename TT>
__host__ cudaMatrix::~cudaMatrix() {deallocate();}

// template<typename TT>
__host__ void cudaMatrix::display()
{
    SynchronizeValues();
    std::cout << " showing top left from Matrix:\n";
    for (int row = 0; row < std::min(MAX_DISPLAY_MATRIX, this->getRow()); row++) {
    for (int col = 0; col < std::min(MAX_DISPLAY_MATRIX, this->getCol()); col++) {
      // Copy old state of cout
      std::ios oldState(nullptr);
      oldState.copyfmt(std::cout);

      // Edit the output format of cout
      std::cout << std::fixed << std::setprecision(2);

      // Print the results
      std::cout << std::setw(8) << this->at(col, row) << " ";

      // Restore the output format of cout
      std::cout.copyfmt(oldState);
    }
    std::cout << std::endl;
  }
}

// template<typename TT>
__host__ data_t& cudaMatrix::data()
{
    return *host_data;
}


// __host__ data_t* cudaMatrix::dataGPU()
// {
//     return dev_data;
// }

__host__ data_t* cudaMatrix::dataGPU()
{
    return (this->dev_data);
}


// template<typename TT>
__host__ data_t& cudaMatrix::at(int x, int y)
{
#ifdef __CUDA_ARCH__
    return dev_data[idx_matrix(_ld, x, y)];
#else
    return host_data[idx_matrix(_ld, x, y)];
#endif
}

// template<typename TT>
__host__ data_t& cudaMatrix::const_at(int x, int y) const
{
#ifdef __CUDA_ARCH__
    return dev_data[idx_matrix(_ld, x, y)];
#else
    return host_data[idx_matrix(_ld, x, y)];
#endif
}


// Returns row value
// template<typename TT>
__host__ int cudaMatrix::getRow() const { return this->_row;}

// Returns Col value 
// template<typename TT>
__host__ int cudaMatrix::getCol() const { return this->_col;}

// Returns Leading Dimension width
// template<typename TT>
__host__ int cudaMatrix::get_ld() const { return this->_ld;}

// template<typename TT>
__host__ int cudaMatrix::getSize() const { return this->getCol() * this->getRow();}

// template<typename TT>
__host__ size_t cudaMatrix::MemSize(bool bin=false) const{
    return get_ld() * getRow() * (bin ? sizeof(data_t) : 1 );
}

//This case really forces to transpose data
// template<typename TT>
__host__ void cudaMatrix::transposeInPlace()
{
    
}

// returns a matrix with the same configuration transposed
// template<typename TT>
__host__ cudaMatrix cudaMatrix::transpose()
{
    cudaMatrix result(this->getCol(), this->getRow());

    
    #pragma omp parallel for
    for(int i=0; i< this->getCol();i++)
        for(int j=0; j< this->getCol();j++)
            result.at(i,j) = this->at(j,i);
    
    return result;

}

//param: out(bool) check if offload executed correctly
// template<typename TT>
/*
    Generate a Random Matrix 
        input: lower_bound, upper_bound [data_t]
        input: Synchronize [bool]
    Synch is used to sync between GPU and CPU
*/      
__host__ void cudaMatrix::randMatrix(data_t lower_bound, data_t upper_bound, bool Synchronize = true)
{
    Change = ChangeOnHost;
    int ld = get_ld();
    for(int ii=0; ii< getRow(); ii++)
    for(int jj=0; jj< getCol(); jj++)
        this->at(ii,jj) = 
            static_cast<data_t>(rand()) / 
            static_cast<data_t>(RAND_MAX / (upper_bound - lower_bound)) + lower_bound ;

    if(Synchronize) SynchronizeValues();
}

void cudaMatrix::changeOccurred(ChangeHandler Status)
{
    this->Change = Status;
}


// template<typename TT>
__host__ bool cudaMatrix::TransferData(hipMemcpyKind kind)
{   
    if(kind == hipMemcpyHostToDevice)
        hipMemcpy(dev_data, host_data, this->MemSize(true), kind);
    else
        hipMemcpy(host_data, dev_data, this->MemSize(true), kind);
    
    check_last_error();
    Change = Equal;
    return true;
}

// template<typename TT>
__host__ void cudaMatrix::SynchronizeValues()
{
    bool sync = false;
    switch(Change){
        case ChangeOnHost: {
            std::cout<< "syn H-> D";
            sync = TransferData(hipMemcpyHostToDevice);break;
        }
        case ChangeOnDevice : {
            std::cout<< "syn D-> H";
            sync = TransferData(hipMemcpyDeviceToHost);break;              
        }
        // default : data already sync. 
    }
    // if didnt sync, keep previous value from change
    this->Change = (sync) ? this->Change : Equal;
}

